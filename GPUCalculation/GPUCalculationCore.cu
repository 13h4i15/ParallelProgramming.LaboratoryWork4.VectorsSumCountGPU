#include "hip/hip_runtime.h"
__global__ void addKernel(int* c, int* a, int* b, unsigned int size)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
	{
		c[index] = a[index] + b[index];
	}
}


#define kernel addKernel
#include "GPUCalculation.h"